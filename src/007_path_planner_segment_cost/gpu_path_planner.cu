#include "hip/hip_runtime.h"
#include "gpu_path_planner.cuh"

__global__ void kernelPathPlanning(double *d_odom, double *d_goal, uint8_t *d_hmap, double *d_cost){
    *d_cost = *(d_odom+1) + *(d_goal+2) + *(d_hmap+3);
}

GPUPathPlanner::GPUPathPlanner(){

}

GPUPathPlanner::~GPUPathPlanner(){
    gpuFree();
}

void GPUPathPlanner::gpuSetup(uint32_t hmap_rows, uint32_t hmap_cols){
    this->hmap_rows = hmap_rows;
    this->hmap_cols = hmap_cols;
    gpuErrchk( hipMalloc((void**)&d_odom, 3*sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&d_goal, 3*sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&d_hmap, hmap_rows*hmap_cols*sizeof(uint8_t)) );
    gpuErrchk( hipMalloc((void**)&d_cost, 1*sizeof(double)) );
}

void GPUPathPlanner::gpuCopyInputToDevice(double *&h_odom, double *&h_goal, uint8_t *&h_hmap){
    gpuErrchk( hipMemcpy(d_odom, h_odom, 3*sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_goal, h_goal, 3*sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_hmap, h_hmap, hmap_rows*hmap_cols*sizeof(uint8_t), hipMemcpyHostToDevice) );
}

void GPUPathPlanner::gpuExecuteKernel(){
    kernelPathPlanning<<<1,1>>>(d_odom, d_goal, d_hmap, d_cost);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}


void GPUPathPlanner::gpuCopyOutputToHost(double *&h_cost){
    gpuErrchk( hipMemcpy(h_cost, d_cost, 1*sizeof(double), hipMemcpyDeviceToHost) );
}

void GPUPathPlanner::gpuFree(){
    gpuErrchk( hipFree(d_odom) );
    gpuErrchk( hipFree(d_goal) );
    gpuErrchk( hipFree(d_hmap) );
    gpuErrchk( hipFree(d_cost) );
}
