#include "hip/hip_runtime.h"
#include "gpu_path_planner.cuh"

#define MAX_SUBEP_LEN 16

__global__ void kernelPathPlanning(double *d_odom, double *d_goal, uint8_t *d_hmap, double *d_cost){
    // int tid = blockIdx.x*blockDim.x + threadIdx.x;

    // Divide episode to subepisodes
    double ep_len = sqrt((d_odom[0] - d_goal[0])*(d_odom[0] - d_goal[0]) + (d_odom[1] - d_goal[1])*(d_odom[1] - d_goal[1]));
    int ep_num = (int)floor(ep_len/MAX_SUBEP_LEN);

    double sin_alfa = (d_goal[1] - d_odom[1]) / ep_len;
    double cos_alfa = (d_goal[0] - d_odom[0]) / ep_len;

    double ep_dx = ep_len / ep_num * cos_alfa;
    double ep_dy = ep_len / ep_num * sin_alfa;


    for(int ep_no = 1; ep_no < ep_num; ep_no++){

        for(int y = 0; y < 8; y++){
            for(int x = 0; x < 8; x++){

                int sub_y = (int)(d_odom[1] + ep_no * ep_dy) - 4 + y;
                int sub_x = (int)(d_odom[0] + ep_no * ep_dx) - 4 + x;

                d_hmap[sub_y * 256 + sub_x] = 0;
            }
        }

        d_hmap[0] = 0;
        d_hmap[1] = 1;
        d_hmap[2] = 2;

    }

    *d_cost = d_odom[0];

    // *d_cost = *(d_odom+1) + *(d_goal+2) + *(d_hmap+3);
}

GPUPathPlanner::GPUPathPlanner(){

}

GPUPathPlanner::~GPUPathPlanner(){
    gpuFree();
}

void GPUPathPlanner::gpuSetup(uint32_t hmap_rows, uint32_t hmap_cols){
    this->hmap_rows = hmap_rows;
    this->hmap_cols = hmap_cols;
    gpuErrchk( hipMalloc((void**)&d_odom, 3*sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&d_goal, 3*sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&d_hmap, hmap_rows*hmap_cols*sizeof(uint8_t)) );
    gpuErrchk( hipMalloc((void**)&d_cost, 1*sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

}

void GPUPathPlanner::gpuCopyInputToDevice(double *h_odom, double *h_goal, uint8_t *h_hmap){
    gpuErrchk( hipMemcpy(d_odom, h_odom, 3*sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_goal, h_goal, 3*sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_hmap, h_hmap, hmap_rows*hmap_cols*sizeof(uint8_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipDeviceSynchronize() );
}

void GPUPathPlanner::gpuExecuteKernel(){
    kernelPathPlanning<<<1,1>>>(d_odom, d_goal, d_hmap, d_cost);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}


void GPUPathPlanner::gpuCopyOutputToHost(double *h_cost, uint8_t *h_hmap){
    gpuErrchk( hipMemcpy(h_cost, d_cost, 1*sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_hmap, d_hmap, hmap_rows*hmap_cols*sizeof(uint8_t), hipMemcpyDeviceToHost) );
    gpuErrchk( hipDeviceSynchronize() );
}

void GPUPathPlanner::gpuFree(){
    gpuErrchk( hipFree(d_odom) );
    gpuErrchk( hipFree(d_goal) );
    gpuErrchk( hipFree(d_hmap) );
    gpuErrchk( hipFree(d_cost) );
}
