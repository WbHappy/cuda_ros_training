#include "hip/hip_runtime.h"
#include <ros/ros.h>
#include <hip/hip_runtime.h>

__global__ void mykernel(int* device_a){
    *device_a = 22;
}

int main(int argc, char** argv){
    ros::init(argc, argv, "hello_world");
    ros::NodeHandle nh;

    int* host_a = (int*) malloc(sizeof(int));
    int* device_a; hipMalloc((void**) &device_a, sizeof(int));
    hipMemcpy(device_a, host_a, sizeof(int), hipMemcpyHostToDevice);
    mykernel<<<1,1>>>(device_a);
    hipMemcpy(host_a, device_a, sizeof(int), hipMemcpyDeviceToHost);

    ROS_INFO("Hello World! %d", *host_a);

    hipFree(device_a);
    free(host_a);
    return 0;
}
