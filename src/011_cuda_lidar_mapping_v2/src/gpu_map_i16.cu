#include "hip/hip_runtime.h"
#include "../include/gpu_map_i16.cuh"

__global__ void fillValueKernel(int16_t* data, int16_t fill_value)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;        // Globalny index x (kolumna na mapie wysokości)
    int idy = blockDim.y * blockIdx.y + threadIdx.y;        // Globalny index y (wiersz na mapie wysokości)
    int tid = idx + idy * gridDim.x * blockDim.x;           // Globalny numer indeksu (adres pamięci na mapie wysokości)

    data[tid] = fill_value;
}


GpuMapI16::GpuMapI16(int size_x, int size_y)
{
    this->resize(size_x, size_y);
}


GpuMapI16::GpuMapI16(int size_x, int size_y, const int16_t fill_value)
{
    this->resize(size_x, size_y);
    this->fill(fill_value);
}


void GpuMapI16::allocate(int size_x, int size_y)
{
    this->size_x = size_x;
    this->size_y = size_y;

    gpuErrchk( hipMalloc((void**)&this->data, size_x * size_y * sizeof(int16_t)) );
}


void GpuMapI16::resize(int size_x, int size_y)
{
    release();
    allocate(size_x, size_y);
}


void GpuMapI16::fill(const int16_t fill_value)
{
    int block_x = 32;
    int block_y = 32;

    int grid_x = (size_x + block_x - 1) / block_x;
    int grid_y = (size_y + block_y - 1) / block_y;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(block_x, block_y, 1);

    fillValueKernel<<< grid, block >>> (this->data, fill_value);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}


void GpuMapI16::release()
{
    gpuErrchk( hipFree(this->data) );
}
