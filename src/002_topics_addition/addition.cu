#include "hip/hip_runtime.h"
#include "addition.cuh"
#include <stdio.h>

__global__ void addition(int* a, int* b, int* c){
    *c = *a + *b;
}

void setupCuda(int* &D_A, int* &D_B, int* &D_C, int* &A, int* &B, int* &C){
    hipMalloc((void**)&D_A, sizeof(int));
    hipMalloc((void**)&D_B, sizeof(int));
    hipMalloc((void**)&D_C, sizeof(int));

    hipMemcpy(D_A, A, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_C, C, sizeof(int), hipMemcpyHostToDevice);
}

void copyInputToDevice(int* D_A, int* D_B, int* D_C, int* A, int* B, int* C){
    hipMemcpy(D_A, A, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(int), hipMemcpyHostToDevice);
}

void executeKernel(int* D_A, int* D_B, int* D_C, int* A, int* B, int* C){
    addition<<<1,1>>>(D_A, D_B, D_C);
}

void copyOutputToHost(int* D_A, int* D_B, int* D_C, int* A, int* B, int* C){
    hipMemcpy(C, D_C, sizeof(int), hipMemcpyDeviceToHost);
}

void cleanupCuda(int* D_A, int* D_B, int* D_C){
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);
}
